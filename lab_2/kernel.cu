#include "hip/hip_runtime.h"
﻿
#include <iostream> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <>
#include <math.h>
using namespace std;//использовано пространство имен std


__global__ void myKernel(float *A, float *B, float *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n)
		C[i] = A[i] + B[i];
}

// тело основной программы
int main(void) {
	setlocale(LC_ALL, "Russian");

	//переменная для хранения параметров GPU-устройства 
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);

	float *hostA, *hostB, *hostC;
	int N = devProp.maxThreadsPerBlock * 512; //512 - блоков в сетке

	cout << devProp.maxThreadsPerBlock << endl;

	hostA = (float*)malloc(N * sizeof(float));
	hostB = (float*)malloc(N * sizeof(float));
	hostC = (float*)malloc(N * sizeof(float));

	float *deviceA, *deviceB, *deviceC;

	hipMalloc((void**)&deviceA, N * sizeof(float));
	hipMalloc((void**)&deviceB, N * sizeof(float));
	hipMalloc((void**)&deviceC, N * sizeof(float));

	//Инициализация массивов случайными значениями 
	for (int i = 0; i < N; i++)
	{
		hostA[i] = rand();
		hostB[i] = rand();
	}

	hipMemcpy(deviceA, hostA, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, N * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop; // Описываем переменные типа hipEvent_t
	float gpuTime = 0.0f;
	hipEventCreate(&start); // Создаём событие начала выполнения ядра
	hipEventCreate(&stop); // Создаём событие конца выполнения ядра 
	hipEventRecord(start, 0); //Привязываем событие start к текущему месту 

	myKernel << <512*4, devProp.maxThreadsPerBlock/4 >> > (deviceA, deviceB, deviceC, N);

	hipEventRecord(stop, 0); //Привязываем событие stop к текущему месту
	
	hipEventSynchronize(stop); //Ждем реального окончания выполнения ядра, используя 
								//возможность синхронизации по событию stop
	
	hipEventElapsedTime(&gpuTime, start, stop); // Запрос времени между событиями start 
												 // и stop
	
	cout << " Время работы мс: " << gpuTime << endl; // Печатаем время

	hipEventDestroy(start); // Уничтожаем событие start hipEventDestroy(stop); 
							 // Уничтожаем событие stop

	hipMemcpy(hostC, deviceC, N * sizeof(float), hipMemcpyDeviceToHost);

	// Вывод на экран результата вычислений
	for (int i = 0; i < 10; i++) {
		cout << hostA[i] << "+" << hostB[i] << "=" << hostC[i] << endl;
	}

	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	free(hostA);
	free(hostB);
	free(hostC);

	//ждем нажатие клавиши 
	system("pause");
	return 0;
}